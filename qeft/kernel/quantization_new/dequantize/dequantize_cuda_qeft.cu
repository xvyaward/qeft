#include "hip/hip_runtime.h"
/*
 * Modified from NVIDIA [TRT-LLM](https://github.com/NVIDIA/TensorRT-LLM/tree/d37b507f41a87457fe9f10f7459d08f5db235745/cpp/tensorrt_llm/kernels/weightOnlyBatchedGemv)
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
@article{lin2023awq,
  title={AWQ: Activation-aware Weight Quantization for LLM Compression and Acceleration},
  author={Lin, Ji and Tang, Jiaming and Tang, Haotian and Yang, Shang and Dang, Xingyu and Han, Song},
  journal={arXiv},
  year={2023}
}
*/

#include <hip/hip_fp16.h>
#include <stdio.h>
#include <torch/extension.h>
#include "dequantize_cuda.h"
#include "../dequantize.cuh"
#define PACK_FACTOR 8
#define WARP_SIZE 32
#define MEM_ACCESS_SIZE 128
#define HALF_SIZE 16

template <int NPerBlock, int BlockSize>
__global__ void dequant_kernel(
  const uint32_t* weight, const half* scales, const half* zeros, half* outputs, 
  const int IC, const int OC, int GroupSize)
{
    const int kStride = 64;
    const int kElemsPerThread = MEM_ACCESS_SIZE / 4;
    const int kThreadsNumPerTile = kStride / kElemsPerThread;
    const int Elt_per_write = MEM_ACCESS_SIZE / HALF_SIZE; // 8
    // assert(MEM_ACCESS_SIZE == 128);

    static constexpr int kShuffleSize = 32;
    static constexpr int kShuffleBasicTile = 2;
    static constexpr int kShuffleContinous = 4;
    static constexpr int kShuffleStrided = 4;

    constexpr int kInterleave = 4;

    uint32_t local_qweights[MEM_ACCESS_SIZE / 32];
    half half_weight_buffer[kElemsPerThread]; 
    half2 local_scale[NPerBlock];
    half2 local_scaled_zeros[NPerBlock];
    
    const int blk_row_offset = blockIdx.x * NPerBlock * kInterleave;
    const int thd_row_offset = (threadIdx.x / kThreadsNumPerTile) % kInterleave;
    const int act_k_offset = threadIdx.x / (kThreadsNumPerTile * kInterleave) * kStride
                               + (threadIdx.x % kThreadsNumPerTile) * kElemsPerThread;
    const int group_offset = act_k_offset / GroupSize;
    
    const uint32_t* blk_weight_ptr = weight + blk_row_offset * IC / PACK_FACTOR;
    const half* scale_ptr = scales + blk_row_offset + thd_row_offset + group_offset * OC;
    const half* zeros_ptr = zeros + blk_row_offset + thd_row_offset + group_offset * OC;

    const int act_forward_step = BlockSize * kElemsPerThread / kInterleave;
    const int scale_forward_step = act_forward_step / GroupSize * OC;

    // Main loop iteration, each block completes the outputs for several OCs
    for (int kk = threadIdx.x * kElemsPerThread; kk < IC * kInterleave; kk += BlockSize * kElemsPerThread)
    {
        // Load qweight, scales and scaled_zeros
        #pragma unroll
        for (int idx = 0; idx < NPerBlock; ++idx)
        {
            // use float4 to load weights, each thread load 32 int4 numbers (1 x float4, 128 bit)
            *((float4*)(local_qweights)) = 
                *((float4*)(blk_weight_ptr + (idx * kInterleave * IC + kk)/ PACK_FACTOR));
            local_scale[idx] = __half2half2(*(scale_ptr + idx * kInterleave));
            local_scaled_zeros[idx] = __half2half2(*(zeros_ptr + idx * kInterleave));
            
            // Map int4 qweight to fp format 
            #pragma unroll
            for (int i = 0; i < MEM_ACCESS_SIZE / 32; ++i)
            {
                // Converts 32 bits (8 x int4) to 8 fp16
                dequantize_s4_to_fp16x2(*reinterpret_cast<half2 *>(local_qweights + i), reinterpret_cast<uint4 *>(half_weight_buffer + i * PACK_FACTOR));
            }

            // Dequantize (apply s/z) and shuffle elements to match the weight packing format
            #pragma unroll
            for (int i = 0; i < kElemsPerThread / 2; ++i)
            {
                *reinterpret_cast<half2*>(half_weight_buffer + i * 2) 
                    = __hfma2(*reinterpret_cast<half2*>(half_weight_buffer + i * 2), 
                        local_scale[idx], 
                        local_scaled_zeros[idx]);
            }

            // Write to DRAM Tensor
            #pragma unroll
            for (int i = 0; i < kElemsPerThread / Elt_per_write; ++i)
            {
                *((float4*)(outputs + (idx * kInterleave + blk_row_offset + thd_row_offset) * IC \
                                    + act_k_offset + i * Elt_per_write))
                                    = *((float4*)(half_weight_buffer + i * Elt_per_write));
            }

        }
        outputs += act_forward_step;
        scale_ptr += scale_forward_step;
        zeros_ptr += scale_forward_step;
    }
}


/*
Args:
  _in_feats: tensor of shape [B, IC];
  _kernel: int tensor of shape [OC, IC // 8];
  _zeros: int tensor of shape [OC, IC // G // 8];
  _scaling_factors: tensor of shape [OC, IC // G];
  blockDim_x: size of thread block, dimension x, where blockDim_x * workload_per_thread = IC;
  blockDim_y: size of thread block, dimension y, where blockDim_y * gridDim_y = OC;

Returns:
  out_feats: tensor of shape [B, OC];
*/
torch::Tensor dequantize_weight_4bit_qeft(
    torch::Tensor _kernel,
    torch::Tensor _scaling_factors,
    torch::Tensor _zeros,
    int n,
    int k,
    int group_size)
{
    auto kernel = reinterpret_cast<uint32_t*>(_kernel.data_ptr());
    auto zeros = reinterpret_cast<half*>(_zeros.data_ptr<at::Half>());
    auto scaling_factors = reinterpret_cast<half*>(_scaling_factors.data_ptr<at::Half>());

    auto options = torch::TensorOptions().dtype(_scaling_factors.dtype()).device(_scaling_factors.device());
    torch::Tensor _half_weight = torch::empty({n,k}, options);
    half * half_weight = reinterpret_cast<half *>(_half_weight.data_ptr());
    
    static constexpr int N_PER_BLOCK = 2;
    static constexpr int K_INTERLEAVE = 4;
    static constexpr int BLOCK_SIZE = 256;

    dim3 num_blocks(n / N_PER_BLOCK / K_INTERLEAVE);
    dim3 num_threads(BLOCK_SIZE);

    dequant_kernel<N_PER_BLOCK, BLOCK_SIZE><<<num_blocks, num_threads>>>(
      kernel, scaling_factors, zeros, half_weight, k, n, group_size
    );
    return _half_weight;
}
